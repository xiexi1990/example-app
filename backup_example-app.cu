
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__ void sum(float *x)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  float mul_temp[32];
  float result = 0;

  // #pragma unroll
  //   for (int j = 0; j < 32; ++j)
  //   {
  //     mul_temp[j] = x[j];
  //   }
  // #pragma unroll
  //   for (int j = 0; j < 32; ++j)
  //   {
  //     result += mul_temp[j];
  //   }
  float xi = x[tid];
  for (int offset = 1; offset < 32; offset *= 2)
  {
    result += __shfl_xor_sync(-1, xi, offset);
  }

  x[0] = result;
}
int main(void)
{
  int N = 32;
  float *x; // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));

  float s = 0;
  for (int i = 0; i < N; i++)
  {
    x[i] = i;
    s += x[i];
  }
  // Run kernel on 1M elements on the GPU
  sum<<<1, 32>>>(x);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  std::cout << "sum result: " << x[0] << " " << s << std::endl;
  // Free memory
  hipFree(x);
  return 0;
}